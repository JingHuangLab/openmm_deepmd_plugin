#include "hip/hip_runtime.h"
extern "C" __global__
void addForces(const FORCES_TYPE* __restrict__ forces, long long* __restrict__ forceBuffers, int* __restrict__ atomIndex, int numAtoms, int paddedNumAtoms) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        forceBuffers[atom] += (long long) (forces[3*index]*0x100000000);
        forceBuffers[atom+paddedNumAtoms] += (long long) (forces[3*index+1]*0x100000000);
        forceBuffers[atom+2*paddedNumAtoms] += (long long) (forces[3*index+2]*0x100000000);
    }
}

